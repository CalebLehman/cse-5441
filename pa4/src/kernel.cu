#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "amr.h"
#include "common.h"
#include "kernel.h"
}

extern "C" {
void launch_kernel(float affect_rate, float epsilon, int num_blocks, int num_thread_pb, BoxData* boxes, DSV* current_vals, DSV* updated_vals, Count N, Count* iter) {
    dim3 dim_grid(num_blocks);
    dim3 dim_block(num_thread_pb);
    kernel<<<dim_grid,dim_block>>>(affect_rate, epsilon, boxes, current_vals, updated_vals, N, iter);
}
}

__global__ void kernel(float affect_rate, float epsilon, BoxData* boxes, DSV* current_vals, DSV* updated_vals, Count N, Count* iter_TODO) {
    AMRMaxMin max_min = getMaxMin(current_vals, N);
    unsigned long iter;
    for (iter = 0; (max_min.max - max_min.min) / max_min.max > epsilon; ++iter, max_min = getMaxMin(current_vals, N)) {
        /**
         * For each box
         */
        for (int i = 0; i < N; ++i) {
            BoxData* box = &boxes[i];
            /**
             * Compute updated DSV
             */
            updated_vals[i] = box->self_overlap * current_vals[i];
            for (int nhbr = 0; nhbr < box->num_nhbrs; ++nhbr) {
                updated_vals[i] += box->overlaps[nhbr] * current_vals[box->nhbr_ids[nhbr]];
            }
            updated_vals[i] /= box->perimeter;
            updated_vals[i] = current_vals[i] * (1 - affect_rate)
                + updated_vals[i] * affect_rate;
        }
        /**
         * Commit updated DSVs
         */
        DSV* temp = current_vals;
        current_vals = updated_vals;
        updated_vals = temp;
    }
}
