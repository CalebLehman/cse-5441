#include "hip/hip_runtime.h"
#include <stdio.h> // TODO
#include <unistd.h> // TODO
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "amr.h"
#include "common.h"
#include "kernel.h"
}

extern "C" {
void launch_kernel(
    float affect_rate,
    float epsilon,
    int num_blocks,
    int num_thread_pb,
    BoxData* boxes,
    DSV* current_vals,
    DSV* updated_vals,
    Count N,
    unsigned long* h_iter
) {
    dim3 temp_dim_grid(1);
    dim3 temp_dim_block(1);

    dim3 dim_grid(num_blocks);
    dim3 dim_block(num_thread_pb);

    AMRMaxMin max_min;
    AMRMaxMin* d_max_min;
    hipMalloc((void**)&d_max_min, sizeof(*d_max_min));
    maxMinKernel<<<temp_dim_grid,temp_dim_block>>>(d_max_min, current_vals, N);
    hipMemcpy(&max_min, d_max_min, sizeof(*d_max_min), hipMemcpyDeviceToHost);

    unsigned long iter;
    for (
        iter = 0;
        (max_min.max - max_min.min) / max_min.max > epsilon;
        ++iter
    ) {
        kernel<<<dim_grid,dim_block>>>(affect_rate, epsilon, boxes, current_vals, updated_vals, N, h_iter);
        hipDeviceSynchronize();


        /**
         * Commit updated DSVs
         */
        DSV* temp = current_vals;
        current_vals = updated_vals;
        updated_vals = temp;

        maxMinKernel<<<temp_dim_grid,temp_dim_block>>>(d_max_min, current_vals, N);
        hipMemcpy(&max_min, d_max_min, sizeof(*d_max_min), hipMemcpyDeviceToHost);
    }

    hipMemcpy(h_iter, &iter, sizeof(*h_iter), hipMemcpyHostToDevice);
}
}

__global__ void maxMinKernel(
    AMRMaxMin* max_min,
    DSV* vals,
    Count N
) {
    *max_min = getMaxMin(vals, N);
}

__global__ void kernel(
    float affect_rate,
    float epsilon,
    BoxData* boxes,
    DSV* current_vals,
    DSV* updated_vals,
    Count N,
    unsigned long* h_iter
) {
    int tid         = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    Count start     = tid * (N / num_threads);
    Count end       = (tid == num_threads - 1)
        ? N
        : (tid + 1) * (N / num_threads);

    /**
     * For each box
     */
    for (int i = start; i < end; ++i) {
        BoxData* box = &boxes[i];
        /**
         * Compute updated DSV
         */
        updated_vals[i] = box->self_overlap * current_vals[i];
        for (int nhbr = 0; nhbr < box->num_nhbrs; ++nhbr) {
            updated_vals[i] +=
                box->overlaps[nhbr] * current_vals[box->nhbr_ids[nhbr]];
        }
        updated_vals[i] /= box->perimeter;
        updated_vals[i] = current_vals[i] * (1 - affect_rate)
            + updated_vals[i] * affect_rate;
    }
}
