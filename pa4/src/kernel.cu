#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "amr.h"
#include "common.h"
#include "kernel.h"
}

extern "C" {
void launch_kernel(int num_blocks, int num_thread_pb, BoxData* boxes, DSV* current_vals, DSV* updated_vals, Count N, Count* iter) {
    dim3 dim_grid(num_blocks);
    dim3 dim_block(num_thread_pb);
    kernel<<<dim_grid,dim_block>>>(boxes, current_vals, updated_vals, N, iter);
}
}

__global__ void kernel(BoxData* boxes, DSV* current_vals, DSV* updated_vals, Count N, Count* iter) {
    #if 0
    unsigned long iter;
    for (iter = 0; (max_min.max - max_min.min) / max_min.max > epsilon; ++iter, max_min = getMaxMin(input)) {
        /**
         * For each box
         */
        for (int i = 0; i < input->N; ++i) {
            BoxData* box = &input->boxes[i];
            /**
             * Compute updated DSV
             */
            updated_vals[i] = box->self_overlap * input->vals[i];
            for (int nhbr = 0; nhbr < box->num_nhbrs; ++nhbr) {
                updated_vals[i] += box->overlaps[nhbr] * input->vals[box->nhbr_ids[nhbr]];
            }
            updated_vals[i] /= box->perimeter;
            updated_vals[i] = input->vals[i] * (1 - affect_rate)
                + updated_vals[i] * affect_rate;
        }

        /**
         * Commit updated DSVs
         */
        DSV* temp = input->vals;
        input->vals = updated_vals;
        updated_vals = temp;
    }
    #endif
}
