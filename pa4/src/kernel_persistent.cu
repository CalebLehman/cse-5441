#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "amr.h"
#include "common.h"
#include "kernel.h"
}

extern "C" {
void launch_kernel(
    float affect_rate,
    float epsilon,
    BoxData* boxes,
    DSV* current_vals,
    DSV* updated_vals,
    Count N,
    unsigned long* h_iter
) {
    dim3 dim_grid(1);
    dim3 dim_block(320);  // Hardcoded optimal value
    kernel<<<dim_grid,dim_block>>>(affect_rate, epsilon, boxes, current_vals, updated_vals, N, h_iter);
}
}

__global__ void kernel(
    float affect_rate,
    float epsilon,
    BoxData* boxes,
    DSV* current_vals,
    DSV* updated_vals,
    Count N,
    unsigned long* h_iter
) {
    int tid         = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    Count start     = tid * (N / num_threads);
    Count end       = (tid == num_threads - 1)
        ? N
        : (tid + 1) * (N / num_threads);

    unsigned long iter;
    AMRMaxMin max_min = getMaxMin(current_vals, N);
    for (
        iter = 0;
        (max_min.max - max_min.min) / max_min.max > epsilon;
        ++iter, max_min = getMaxMin(current_vals, N)
    ) {
        /**
         * For each box
         */
        for (int i = start; i < end; ++i) {
            BoxData* box = &boxes[i];
            /**
             * Compute updated DSV
             */
            updated_vals[i] = box->self_overlap * current_vals[i];
            for (int nhbr = 0; nhbr < box->num_nhbrs; ++nhbr) {
                updated_vals[i] +=
                    box->overlaps[nhbr] * current_vals[box->nhbr_ids[nhbr]];
            }
            updated_vals[i] /= box->perimeter;
            updated_vals[i] = current_vals[i] * (1 - affect_rate)
                + updated_vals[i] * affect_rate;
        }
        /**
         * Commit updated DSVs
         */
        DSV* temp = current_vals;
        current_vals = updated_vals;
        updated_vals = temp;

        __syncthreads();
    }

    if (tid == 0) {
        *h_iter = iter;
    }
}
